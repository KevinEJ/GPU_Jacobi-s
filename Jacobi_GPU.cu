#include "hip/hip_runtime.h"
#include <iostream>
#include <sstream>
#include <fstream>
#include <string>
#include <vector>
#include <ctime>

#include "util.h"
#include "J_kernel.h"

int main(int argc, char *argv[]){
    
    // Parsing input 
    if( argc != 2 )
        cerr << " Usage: ./J_CPU input_n \n" ;
    
    string input_num =  argv[1]  ; 
    string filename = "inputs/" + input_num + ".txt" ; 

    int* n , *iter ;
    float* input ;
    float* sol; 

    n = new int(0) ; 
    iter = new int(0) ; 
    getinput( filename ,n , iter , input , sol ) ; 
    printf( " n = %d \n" , *n ) ;
    
    /*for( int i = 0 ; i < *n ; i++){
        printf( " c[%d] = %f \n" , i , sol[i] ) ;
    }
    for( int i = 0 ; i < *n ; i++){
        for( int j = 0 ; j < *n ; j++)
            printf( "%f , " ,  input[i][j] ) ;
        printf( "\n") ;
    }*/
    
    float* x_k  ; //= new float[*n]; 
    float* x_k1 ; //= new float[*n]; 
    hipMallocManaged(&x_k, (*n)*sizeof(float));
    hipMallocManaged(&x_k1, (*n)*sizeof(float));

    clock_t c_start = clock();
    for ( int it = 0 ; it < *iter ; it++){
        printf( "iter = %d \n" , it ) ;
        J_kernel<<< 100 , 100  >>> ( *n , input , sol , x_k , x_k1 ) ;
        hipDeviceSynchronize(); 
        float* temp ; 
        temp = x_k ; 
        x_k = x_k1 ;
        x_k1 = temp ; 
        //J_kernel<<< 100 , 100  >>> ( *n , input , sol , x_k1 , x_k ) ;
        //hipDeviceSynchronize(); 
        for( int i = 0 ; i < *n ; i++){
        //    printf( " x_k1[%d] = %f \n" , i , x_k1[i] ) ;
        }
        for( int i = 0 ; i < *n ; i++){
        //    printf( " x_k[%d] = %f \n" , i , x_k[i] ) ;
        }
    }
    hipDeviceSynchronize(); 

/*    for ( int it = 0 ; it < *iter ; it++ ){
        printf( "iter = %d \n" , it ) ;
        for ( int i = 0 ; i < *n ; i ++ ){
            float t = 0 ; 
            for( int j = 0 ; j < *n ; j ++){
                if ( i!=j ){
                    t += input[i][j]*x_k[j] ; 
                }
            }
            x_k1[i] = ( sol[i] - t ) / input[i][i] ; 
        }
        float* temp ; 
        temp = x_k ; 
        x_k = x_k1 ;
        x_k1 = temp ; 
    }
*/  
    clock_t c_end = clock();

    for( int i = 0 ; i < *n ; i++){
        printf( " x[%d] = %f \n" , i , x_k[i] ) ;
    }

    float* res = MatrixMultiple( input , x_k , *n) ;  
    for( int i = 0 ; i < *n ; i++){
        printf( " res[%d] = %f    |  sol[%d] = %f  \n" , i , res[i] , i , sol[i] ) ;
    }

    double time_elapsed_ms = 1000.0 * (c_end-c_start) / CLOCKS_PER_SEC;
    cout << "CPU time used: " << time_elapsed_ms/1000.0 << " ms\n";
    return 0 ;
}

/*
GPU 
Naive : 
    for( k iteratinos )
        kernel<<<>>>
            for( all a[thread.x][j] ) 
                t += a * x_old
            x_new[thread.x] = ( b[ thread.x ] - t ) / a[i][i] ; 
Shared Memory 

*/


