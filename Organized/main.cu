#include "hip/hip_runtime.h"

#include <iostream>
#include <string>
#include <ctime>

#include "util.h"
#include "CPU_Jacobi.h"
#include "GPU_Unified.h"
#include "GPU_MemCopy.h"
#include "GPU_Shared.h"
//#include "GPU_Stream.h"
#include "GPU_MemCoa.h"
#include "GPU_Reduce.h"
#include "GPU_Reduce2.h"
#include "GPU_Reduce3.h"


double exeTime ; 
int    g_Block_size ; 

int main(int argc, char *argv[]){

    // Parsing input 
    if( argc != 3 )
        cerr << " Usage: ./Jacobi input_n mode\n" ;
    
    int mode = stoi(argv[2]) ; // 0: CPU , 1: Unified, 2: Memcopy, 3:Shared, 4:....
    //g_Block_size = stoi(argv[3]) ; // 0: CPU , 1: Unified, 2: Memcopy, 3:Shared, 4:....
    string input_num =  argv[1]  ; 
    string filename = "inputs/" + input_num + ".txt" ; 

    //Declare General Variables
    int n , iter ;
    float* input ;
    float *sol , *x_k , *x_k1 ; 

    getinput( filename , n , iter , input , sol ) ; 
    printf( " n = %d \n" , n ) ;

    x_k  = new float[n] ; 
    x_k1 = new float[n] ; 
  
    for( int i = 0 ; i < n ; i++)
        x_k[i] = 0 ;
    
    // Tans input 
    float* t_input = new float[ n*n ] ; 
    for( int i = 0 ; i < n ; i++){
        for( int j = 0 ; j < n ; j ++ ){
            t_input[ j*n + i ]  = input[ i*n + j ] ; 
        }
    }

    //Implements 
    clock_t c_start = clock();
    //Mem copy 
    //clock_t c_mem_start = clock();
    if( mode == 0 )
        CPU_Jacobi( n , iter , input , sol , x_k , x_k1 ) ; 
    else if( mode == 1 )
        GPU_Unified( n , iter , input , sol , x_k , x_k1 ) ; 
    else if( mode == 2 )
        GPU_MemCopy( n , iter , input , sol , x_k , x_k1 ) ; 
    else if( mode == 3 )
        GPU_Shared( n , iter , t_input , sol , x_k , x_k1 ) ; 
    else if( mode == 4 )
        GPU_Memcoalesc( n , iter , t_input , sol , x_k , x_k1 ) ; 
    else if( mode == 5 )
        GPU_Reduction( n , iter , input , sol , x_k , x_k1 ) ; 
    else if( mode == 6 )
        GPU_Reduction2( n , iter , input , sol , x_k , x_k1 ) ; 
    else if( mode == 7 )
        GPU_Reduction3( n , iter , input , sol , x_k , x_k1 ) ; 
    //clock_t c_mem_end = clock();

    //Kernel Call
    //clock_t c_exe_start = clock();
    
    //clock_t c_exe_end = clock();

    //Mem copy back 
    //clock_t c_memback_start = clock();
    //clock_t c_memback_end = clock();
    clock_t c_end = clock();

    //Verification
    float* res = MatrixMultiple( input , x_k , n) ; 
    bool check = true ; 
    //print_1D_array( n , "x" , x_k ) ;
    for( int i = 0 ; i < n ; i++){
        if( abs(res[i]-sol[i]) > 1){
            printf( "Answer is wrong !! \n" );
            check = false;
        }
    //    printf( " res[%d] = %f    |  sol[%d] = %f  \n" , i , res[i] , i , sol[i] ) ;
    }
    if( check ){
        printf( "Answer is correct \n");
    }

    hipDeviceSynchronize();
    //Delete 
    delete[] input ; 
    delete[] sol ; 
    delete[] x_k ; 
    delete[] x_k1 ; 
    delete[] res ; 
    /*  
    double memcopy_time = 1000.0 * (c_mem_end-c_mem_start) / CLOCKS_PER_SEC;
    cout << "Memcopy time used: " << memback_time/1000.0 << " s\n";
    double exe_time     = 1000.0 * (c_exe_end - c_exe_start) / CLOCKS_PER_SEC;
    cout << "Execute time used: " << exe_time    /1000.0 << " s\n";
    double memback_time = 1000.0 * (c_memback_end-c_memback_start) / CLOCKS_PER_SEC;
    cout << "Memback time used: " << memback_time/1000.0 << " s\n";
    */
    double time_elapsed_ms = 1000.0 * (c_end-c_start) / CLOCKS_PER_SEC;
    cout << "Total  time used: " << time_elapsed_ms/1000.0 << " s\n";
    cout << "Kernel time used: " << exeTime/1000.0 << " s\n";
    return 0; 
}
