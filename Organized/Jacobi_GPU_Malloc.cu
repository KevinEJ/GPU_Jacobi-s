#include "hip/hip_runtime.h"
#include <iostream>
#include <sstream>
#include <fstream>
#include <string>
#include <vector>
#include <ctime>

#include "util.h"
#include "J_kernel.h"

int main(int argc, char *argv[]){
    
    // Parsing input 
    if( argc != 2 )
        cerr << " Usage: ./J_CPU input_n \n" ;
    
    string input_num =  argv[1]  ; 
    string filename = "inputs/" + input_num + ".txt" ; 

    int* n , *iter ;
    float* input ;
    float* sol; 

    n = new int(0) ; 
    iter = new int(0) ; 
    getinput( filename ,n , iter , input , sol ) ; 
    printf( " n = %d \n" , *n ) ;
    
    /*for( int i = 0 ; i < *n ; i++){
        printf( " c[%d] = %f \n" , i , sol[i] ) ;
    }
    for( int i = 0 ; i < *n ; i++){
        for( int j = 0 ; j < *n ; j++)
            printf( "%f , " ,  input[ i*(*n) + j] ) ;
        printf( "\n") ;
    }*/
    
    float* x_k  =  new float[*n]; 
    //float* x_k1 =  new float[*n]; 
    float* d_x_k  ; //= new float[*n]; 
    float* d_x_k1 ; //= new float[*n]; 
    
    
    for( int i = 0 ; i < *n ; i++){
        x_k[i] = 0 ;
    }
    hipMalloc((void**)&d_x_k , (*n)*sizeof(float)) ; 
    hipMalloc((void**)&d_x_k1, (*n)*sizeof(float)) ; 
    hipMemcpy( d_x_k , x_k , (*n)*sizeof(float) , hipMemcpyHostToDevice);
    //hipMemcpy( d_x_k1, &x_k1, (*n)*sizeof(float) , hipMemcpyHostToDevice);
    float *d_input, *d_sol ; 
    hipMalloc((void**)&d_input  , (*n)*(*n)*sizeof(float)) ; 
    hipMalloc((void**)&d_sol    , (*n)*sizeof(float)) ; 
    hipMemcpy( d_input , input , (*n)*(*n)*sizeof(float) , hipMemcpyHostToDevice);
    hipMemcpy( d_sol   , sol   , (*n)*sizeof(float) , hipMemcpyHostToDevice);

    clock_t c_start = clock();
    //for ( int it = 0 ; it < *iter ; it++){
    hipStream_t streams[1024] ; 
    
    for( int i  = 0 ; i < 1024 ; i++){
        hipStreamCreate( &streams[i] );        
    }
    
    for ( int it = 0 ; it < *iter ; it++){
        printf( "iter = %d \n" , it ) ;
        //J_kernel<<< 32 , 32  >>> ( *n , d_input , d_sol , d_x_k , d_x_k1 ) ;
        J_kernel<<< 80 , 64  >>> ( *n , 320 , d_input , d_sol , d_x_k , d_x_k1 ) ;
        //J_kernel<<< 64 , 80  >>> ( *n , 64 , d_input , d_sol , d_x_k , d_x_k1 ) ;
        hipDeviceSynchronize(); 
        float* temp ; 
        temp = d_x_k ; 
        d_x_k = d_x_k1 ;
        d_x_k1 = temp ; 
    }
    hipDeviceSynchronize(); 
    printf( "finish Jacobi \n") ;
/*    for ( int it = 0 ; it < *iter ; it++ ){
        printf( "iter = %d \n" , it ) ;
        for ( int i = 0 ; i < *n ; i ++ ){
            float t = 0 ; 
            for( int j = 0 ; j < *n ; j ++){
                if ( i!=j ){
                    t += input[i][j]*x_k[j] ; 
                }
            }
            x_k1[i] = ( sol[i] - t ) / input[i][i] ; 
        }
        float* temp ; 
        temp = x_k ; 
        x_k = x_k1 ;
        x_k1 = temp ; 
    }
*/  
    clock_t c_end = clock();

    hipMemcpy( x_k   , d_x_k   , (*n)*sizeof(float) , hipMemcpyDeviceToHost);
    
    for( int i = 0 ; i < *n ; i++){
        printf( " x[%d] = %f \n" , i , x_k[i] ) ;
    }

    float* res = MatrixMultiple( input , x_k , *n) ;  
    for( int i = 0 ; i < *n ; i++){
        printf( " res[%d] = %f    |  sol[%d] = %f  \n" , i , res[i] , i , sol[i] ) ;
    }

    double time_elapsed_ms = 1000.0 * (c_end-c_start) / CLOCKS_PER_SEC;
    cout << "CPU time used: " << time_elapsed_ms/1000.0 << " ms\n";

    hipFree(d_input); 
    hipFree(d_sol); 
    hipFree(d_x_k); 
    hipFree(d_x_k1);
    delete[] input;
    delete[] sol;
    delete[] x_k;
    delete n ;
    delete iter ;

    return 0 ;
}

/*
GPU 
Naive : 
    for( k iteratinos )
        kernel<<<>>>
            for( all a[thread.x][j] ) 
                t += a * x_old
            x_new[thread.x] = ( b[ thread.x ] - t ) / a[i][i] ; 
Shared Memory 

*/


